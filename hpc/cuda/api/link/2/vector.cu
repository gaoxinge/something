#include "hip/hip_runtime.h"
#include "vector.h"
#define N 2048
#define M 512

__global__ void cuda_vector_add(int *a, int *b, int *c, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) c[index] = a[index] + b[index];
}

void vector_add(int *a, int *b, int *c, int n) {
    int *d_a, *d_b, *d_c;
    

    hipMalloc((void **) &d_a, n * sizeof(int));
    hipMalloc((void **) &d_b, n * sizeof(int));
    hipMalloc((void **) &d_c, n * sizeof(int));

    hipMemcpy(d_a, a, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * sizeof(int), hipMemcpyHostToDevice);

    cuda_vector_add<<<(N+M-1)/M,M>>>(d_a, d_b, d_c, n);
    
    hipMemcpy(c, d_c, n * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
}
