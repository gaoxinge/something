
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 512

void random_ints(int *a, int n) {
    for (int i = 0; i < n; i++) {
        // a[i] = 1;
        a[i] = rand();
    }
}

void print_ints(int *a, int n) {
    for (int i = 0; i < n; i++) {
        printf("%d\n", a[i]);
    }
}

__global__ void add(int *a, int *b, int *c) {
    c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

int main() {
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;

    a = (int *) malloc(N * sizeof(int));
    b = (int *) malloc(N * sizeof(int));
    c = (int *) malloc(N * sizeof(int));

    random_ints(a, N);
    random_ints(b, N);

    hipMalloc((void **) &d_a, N * sizeof(int));
    hipMalloc((void **) &d_b, N * sizeof(int));
    hipMalloc((void **) &d_c, N * sizeof(int));

    hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    add<<<N,1>>>(d_a, d_b, d_c);

    hipMemcpy(c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);

    print_ints(c, N);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(a);
    free(b);
    free(c);
    return 0;
}
