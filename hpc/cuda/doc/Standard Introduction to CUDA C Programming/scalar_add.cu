
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void add(int *a, int *b, int *c) {
    *c = *a + *b;
}

int main() {
    int a, b, c;
    int *d_a, *d_b, *d_c;

    a = 2;
    b = 7;

    hipMalloc((void **) &d_a, sizeof(int));
    hipMalloc((void **) &d_b, sizeof(int));
    hipMalloc((void **) &d_c, sizeof(int));

    hipMemcpy(d_a, &a, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, &b, sizeof(int), hipMemcpyHostToDevice);

    add<<<1,1>>>(d_a, d_b, d_c);

    hipMemcpy(&c, d_c, sizeof(int), hipMemcpyDeviceToHost);

    printf("%d\n", c);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c); 
    return 0;
}
