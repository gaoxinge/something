
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define N 1000000

__global__ void vector_add(float *out, float *a, float *b, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;  // blockDim.x = block_size

    if (tid < n) {
        out[tid] = a[tid] + b[tid];
    }
}

int main() {
    float *a, *b, *out;
    float *d_a, *d_b, *d_out;

    a = (float *) malloc(sizeof(float) * N);
    b = (float *) malloc(sizeof(float) * N);
    out = (float *) malloc(sizeof(float) * N);

    for (int i = 0; i < N; i++) {
       a[i] = 1.0f;
       b[i] = 2.0f;
    }

    hipMalloc((void **) &d_a, sizeof(float) * N);
    hipMalloc((void **) &d_b, sizeof(float) * N);
    hipMalloc((void **) &d_out, sizeof(float) * N);

    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);
     
    int block_size = 256;
    int grid_size = N / block_size + (N % block_size != 0 ? 1 : 0);
    vector_add<<<grid_size, block_size>>>(d_out, d_a, d_b, N);

    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);

    // for (int i = 0; i < N; i++) {
    //     printf("%0.1f ", out[i]);
    // }
     
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_out);
    free(a);
    free(b);
    free(out);
    return 0;
}
