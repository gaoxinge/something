#include "hip/hip_runtime.h"
#include "cuda_math.h"


__global__ void cuda_math_sqrt(float *a, float *b) {
    *b = sqrt(*a);
}


float math_sqrt(float a) {
    float b;
    float *d_a,  *d_b;

    hipMalloc((void **) &d_a, sizeof(float));
    hipMalloc((void **) &d_b, sizeof(float));
    
    hipMemcpy(d_a, &a, sizeof(float), hipMemcpyHostToDevice);

    cuda_math_sqrt<<<1,1>>>(d_a, d_b);

    hipMemcpy(&b, d_b, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);

    return b;
}

