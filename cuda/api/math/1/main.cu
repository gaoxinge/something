
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void math_sqrt(float *a, float *b) {
    *b = sqrt(*a);
}

int main() {
    float a, b;
    float *d_a, *d_b;

    a = 4;
    
    hipMalloc((void **) &d_a, sizeof(float));
    hipMalloc((void **) &d_b, sizeof(float));

    hipMemcpy(d_a, &a, sizeof(float), hipMemcpyHostToDevice);

    math_sqrt<<<1, 1>>>(d_a, d_b);

    hipMemcpy(&b, d_b, sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);

    printf("%f\n", b);
    return 0;
}
