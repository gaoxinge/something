
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 2048*2048
#define M 512


__global__ void add(int *a, int *b, int *c, int n) {
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < n) {
        c[index] = a[index] + b[index];
    }
}

void random_ints(int *a, int n) {
    for (int i = 0; i < n; i++) {
        a[i] = rand();
    }
}

void print_ints(int *a, int n) {
    for (int i = 0; i < n; i++) {
        printf("%d\n", a[i]);
    }
}


int main() {
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;

    a = (int *) malloc(N * sizeof(int));
    b = (int *) malloc(N * sizeof(int));
    c = (int *) malloc(N * sizeof(int));

    random_ints(a, N);
    random_ints(b, N);

    hipMalloc((void **) &d_a, N * sizeof(int));
    hipMalloc((void **) &d_b, N * sizeof(int));
    hipMalloc((void **) &d_c, N * sizeof(int));
    
    hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    add<<<(N+M-1)/M,M>>>(d_a, d_b, d_c, N);

    hipMemcpy(c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    
    print_ints(c, N);

    free(a);
    free(b);
    free(c);    
    return 0;
}
