
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 512


__global__ void add(int *a, int *b, int *c) {
    c[threadIdx.x] = a[threadIdx.x] + b[threadIdx.x];
}

void random_ints(int *a, int n) {
    for (int i = 0; i < n; i++) {
        a[i] = rand();
    }
}

void print_ints(int *a, int n) {
    for (int i = 0; i < n; i++) {
        printf("%d\n", a[i]);
    }
}


int main() {
    int *a, *b, *c;
    int *d_a, *d_b, *d_c;

    a = (int *) malloc(N * sizeof(int));
    b = (int *) malloc(N * sizeof(int));
    c = (int *) malloc(N * sizeof(int));

    random_ints(a, N);
    random_ints(b, N);

    hipMalloc((void **) &d_a, N * sizeof(int));
    hipMalloc((void **) &d_b, N * sizeof(int));
    hipMalloc((void **) &d_c, N * sizeof(int));
    
    hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    add<<<1,N>>>(d_a, d_b, d_c);

    hipMemcpy(c, d_c, N * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    
    print_ints(c, N);

    free(a);
    free(b);
    free(c);    
    return 0;
}
